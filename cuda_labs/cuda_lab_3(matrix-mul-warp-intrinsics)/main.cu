#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_cooperative_groups.h"

#include <stdio.h>
#include <iostream>
#include <random>
#include <windows.h>
#include <time.h> 
#include <assert.h>

static const double VALUES_MIN = -1.0;
static const double VALUES_MAX = 1.0;
static const size_t CUDA_BLOCK_SIZE = 32;
static const size_t MATRIX_SIZES_TO_TEST[3] = { 500, 1000, 1500};
enum class MultType { GPU, GPU_SHARED, GPU_WARP_INTRINSICS_1, GPU_WARP_INTRINSICS_2 };

using namespace cooperative_groups;

class MyCudaTimer {
private:
	hipEvent_t start_event, end_event;
public:
	MyCudaTimer() {
		hipEventCreate(&start_event);
		hipEventCreate(&end_event);
	}

	~MyCudaTimer() {
		hipEventDestroy(start_event);
		hipEventDestroy(end_event);
	}

	void start() {
		hipEventRecord(start_event, 0);
	}

	float count_time() {
		float elapsed_time;
		hipEventRecord(end_event, 0);
		hipEventSynchronize(end_event);
		hipEventElapsedTime(&elapsed_time, start_event, end_event);
		return elapsed_time;
	}
};

void fill_matrix_rnd(double* matrix, size_t matrix_size) {

	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<double> distrib(VALUES_MIN, VALUES_MAX);
	size_t element_count = matrix_size * matrix_size;
	double* result = new double[element_count];
	for (size_t i = 0; i < element_count; ++i) {
		matrix[i] = distrib(gen);
	}
	return;
}

template <size_t block_size>
__global__ void mul_on_gpu_warp_intrinsics_kernel_slow_1(double* a, double* b, double* result, size_t matrix_size) {
	thread_block block = this_thread_block();
	dim3 block_index = block.group_index();
	dim3 thread_index = block.thread_index();

	size_t tx = thread_index.x;
	size_t ty = thread_index.y;

	size_t i = block_size * block_index.y + ty;
	size_t j = block_size * block_index.x + tx;

	size_t aj;
	size_t bi;
	double sum = 0.0;

	for (size_t ind = 0; ind * block_size < matrix_size; ind++) {
		aj = tx + block_size * ind;
		bi = ty + block_size * ind;

		double as;
		__shared__ double bs[block_size][block_size];

		as = 0;
		bs[ty][tx] = 0;
		if (i < matrix_size && aj < matrix_size){
			as = a[i * matrix_size + aj];
		}
		if (j < matrix_size && bi < matrix_size){
			bs[ty][tx] = b[bi * matrix_size + j];
		}

		block.sync();

		for (size_t k = 0; k < block_size; k++){
			sum += __shfl_sync(-1, as, k) * bs[k][tx];
		}
		block.sync();
	}

	if (i < matrix_size && j < matrix_size){
		result[i * matrix_size + j] = sum;
	}
}

__global__ void mul_on_gpu_warp_intrinsics_kernel(double* a, double* b, double* result, size_t matrix_size) {

	const size_t block_size = 32;
	const size_t warp_width = 8;
	const size_t warp_height = 4;

	thread_block block = this_thread_block();
	dim3 block_index = block.group_index();
	dim3 thread_index = block.thread_index();


	size_t wid = thread_index.y;
	size_t wy = wid / warp_height;
	size_t wx = wid % warp_height;

	size_t lane = thread_index.x;
	size_t ty = lane / warp_width;
	size_t tx = lane % warp_width;


	size_t i = block_index.y * block_size * 2 + wy * warp_width + ty;
	size_t j = block_index.x * block_size + wx * warp_width + tx;

	size_t aj;
	size_t bi;
	double sum[2] = { 0.0, 0.0 };

	for (size_t ind = 0; ind * warp_width < matrix_size; ind++) {

		aj = tx + warp_width * ind;
		bi = ty + warp_width * ind;

		double as[2];
		double bs[2];

		as[0] = (i < matrix_size&& aj < matrix_size) ? a[i * matrix_size + aj] : 0.0;
		as[1] = ((i + warp_height) < matrix_size && aj < matrix_size) ? a[(i + warp_height) * matrix_size + aj] : 0.0;


		bs[0] = (j < matrix_size&& bi < matrix_size) ? b[bi * matrix_size + j] : 0.0;
		bs[1] = (j < matrix_size && (bi + warp_height) < matrix_size) ? b[(bi + warp_height) * matrix_size + j] : 0.0;

		double b_k_j;
		double a_i_k;
		for (size_t k = 0; k < warp_width; k++) {
			b_k_j = __shfl_sync(-1, bs[k / warp_height], (k % warp_height) * warp_width + tx);

			a_i_k = __shfl_sync(-1, as[0], ty * warp_width + k);
			sum[0] += a_i_k * b_k_j;

			a_i_k = __shfl_sync(-1, as[1], ty * warp_width + k);
			sum[1] += a_i_k * b_k_j;
		}
	}

	if (i < matrix_size && j < matrix_size) {
		result[i * matrix_size + j] = sum[0];
	}
	if ((i + warp_height) < matrix_size && j < matrix_size) {
		result[(i + warp_height) * matrix_size + j] = sum[1];
	}
}


template <size_t block_size>
__global__ void mul_on_gpu_shared_kernel(double* a, double* b, double* result, size_t matrix_size) {

	thread_block block = this_thread_block();
	dim3 block_index = block.group_index();
	dim3 thread_index = block.thread_index();

	size_t tx = thread_index.x;
	size_t ty = thread_index.y;

	size_t i = block_size * block_index.y + ty;
	size_t j = block_size * block_index.x + tx;

	size_t aj;
	size_t bi;
	double sum = 0.0;
	
	for (size_t ind = 0; ind * block_size < matrix_size; ind++) {
		aj = tx + block_size * ind;
		bi = ty + block_size * ind;

		__shared__ double as[block_size][block_size];
		__shared__ double bs[block_size][block_size];

		as[ty][tx] = 0;
		bs[ty][tx] = 0;
		if (i < matrix_size && aj < matrix_size){
			as[ty][tx] = a[i * matrix_size + aj];
		}
		if (j < matrix_size && bi < matrix_size){
			bs[ty][tx] = b[bi * matrix_size + j];
		}

		block.sync();
		for (size_t k = 0; k < block_size; k++)
			sum += as[ty][k] * bs[k][tx];
		block.sync();
	}

	if (i < matrix_size && j < matrix_size){
		result[i * matrix_size + j] = sum;
	}	
}


template <size_t block_size>
__global__ void mul_on_gpu_kernel(double* a, double* b, double* result, size_t matrix_size) {
	thread_block block = this_thread_block();
	dim3 block_index = block.group_index();
	dim3 thread_index = block.thread_index();

	size_t i = block_size * block_index.y + thread_index.y;
	size_t j = block_size * block_index.x + thread_index.x;

	if (i >= matrix_size || j >= matrix_size)
		return;

	size_t ind = i * matrix_size + j;
	result[ind] = 0;

	for (size_t k = 0; k < matrix_size; ++k) {
		result[ind] += a[i * matrix_size + k] * b[k * matrix_size + j];
	}
}


double process_on_gpu(double* matrix_A, double* matrix_B, double* result, size_t matrix_size, MultType mult_type) {
	MyCudaTimer timer;

	double* gpu_mem_A;
	double* gpu_mem_B;
	double* gpu_mem_res;
	size_t bytes_count = matrix_size * matrix_size * sizeof(double);
	hipMalloc((void**)&gpu_mem_A, bytes_count);
	hipMalloc((void**)&gpu_mem_B, bytes_count);
	hipMalloc((void**)&gpu_mem_res, bytes_count);

	hipMemcpy(gpu_mem_A, matrix_A, bytes_count, hipMemcpyHostToDevice);
	hipMemcpy(gpu_mem_B, matrix_B, bytes_count, hipMemcpyHostToDevice);

	dim3 cuda_threads(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);
	dim3 cuda_blocks((matrix_size + cuda_threads.x - 1) / cuda_threads.x, (matrix_size + cuda_threads.y - 1) / cuda_threads.y);
	timer.start();
	switch (mult_type)
	{
	case MultType::GPU:
		mul_on_gpu_kernel<CUDA_BLOCK_SIZE> <<< cuda_blocks, cuda_threads >>> (gpu_mem_A, gpu_mem_B, gpu_mem_res, matrix_size);
		break;
	case MultType::GPU_SHARED:
		mul_on_gpu_shared_kernel<CUDA_BLOCK_SIZE> <<< cuda_blocks, cuda_threads >>> (gpu_mem_A, gpu_mem_B, gpu_mem_res, matrix_size);
		break;
	case MultType::GPU_WARP_INTRINSICS_1:
		mul_on_gpu_warp_intrinsics_kernel_slow_1<CUDA_BLOCK_SIZE> << < cuda_blocks, cuda_threads >> > (gpu_mem_A, gpu_mem_B, gpu_mem_res, matrix_size);
		break;	
	case MultType::GPU_WARP_INTRINSICS_2:
		cuda_threads= dim3(32, 32);
		cuda_blocks = dim3((matrix_size + cuda_threads.x - 1) / cuda_threads.x, (matrix_size + cuda_threads.y * 2  - 1) / (cuda_threads.y * 2));
		mul_on_gpu_warp_intrinsics_kernel << < cuda_blocks, cuda_threads >> > (gpu_mem_A, gpu_mem_B, gpu_mem_res, matrix_size);
		break;
	default:
		return -1;
	}
	

	float elapsed_time = timer.count_time();

	hipMemcpy(result, gpu_mem_res, bytes_count, hipMemcpyDeviceToHost);

	hipFree(gpu_mem_A);
	hipFree(gpu_mem_B);
	hipFree(gpu_mem_res);

	return elapsed_time / 1000.0f;
}

double get_max_diff(double* matrix_A, double* matrix_B, size_t matrix_size) {
	size_t element_count = matrix_size * matrix_size;
	double result = 0.0;
	for (size_t i = 0; i < element_count; ++i) {

		result = std::max(result, std::fabs(matrix_A[i] - matrix_B[i]));
	}

	return result;
}


int main(int argc, char* argv[]) {

	for (size_t matrix_size : MATRIX_SIZES_TO_TEST) {
		double* matrix_A = new double[matrix_size * matrix_size]; 
		double* matrix_B = new double[matrix_size * matrix_size];
		fill_matrix_rnd(matrix_A, matrix_size);
		fill_matrix_rnd(matrix_B, matrix_size);
		double* res_on_gpu_shared = new double[matrix_size * matrix_size];
		double* res_on_gpu_wi_1 = new double[matrix_size * matrix_size];
		double* res_on_gpu_wi_2 = new double[matrix_size * matrix_size];



		float time_on_gpu_shared = process_on_gpu(matrix_A, matrix_B, res_on_gpu_shared, matrix_size, MultType::GPU_SHARED);
		float time_on_gpu_wi_1 = process_on_gpu(matrix_A, matrix_B, res_on_gpu_wi_1, matrix_size, MultType::GPU_WARP_INTRINSICS_1);
		float time_on_gpu_wi_2 = process_on_gpu(matrix_A, matrix_B, res_on_gpu_wi_2, matrix_size, MultType::GPU_WARP_INTRINSICS_2);
		double max_diff_1 = get_max_diff(res_on_gpu_wi_1, res_on_gpu_shared, matrix_size);
		double max_diff_2 = get_max_diff(res_on_gpu_wi_2, res_on_gpu_shared, matrix_size);

		std::cout << "-------------------------------" << std::endl;
		std::cout << "matrix_size: \t" << matrix_size << std::endl;
		std::cout << "time on GPU_shared: \t" << time_on_gpu_shared << std::endl;
		std::cout << "time on GPU_warp_1: \t" << time_on_gpu_wi_1 << std::endl;
		std::cout << "time on GPU_warp_2: \t" << time_on_gpu_wi_2 << std::endl;
		std::cout << "max diff_1: \t" << max_diff_1 << std::endl;
		std::cout << "max diff_2: \t" << max_diff_2 << std::endl;
		std::cout << std::endl;

		delete[] matrix_A;
		delete[] matrix_B;
		delete[] res_on_gpu_shared;
		delete[] res_on_gpu_wi_1;
		delete[] res_on_gpu_wi_2;
	}
	return 0;
}