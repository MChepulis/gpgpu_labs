#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <random>
#include <windows.h>
#include <time.h> 

static const double VALUES_MIN = -1.0;
static const double VALUES_MAX = 1.0;
static const size_t CUDA_BLOCK_SIZE = 32;
static const size_t MATRIX_SIZES_TO_TEST[3] = { 500, 1000, 1500 };
enum class MultType { GPU, GPU_SHARED };


class MyCudaTimer {
private:
	hipEvent_t start_event, end_event;
public:
	MyCudaTimer() {
		hipEventCreate(&start_event);
		hipEventCreate(&end_event);
	}

	~MyCudaTimer() {
		hipEventDestroy(start_event);
		hipEventDestroy(end_event);
	}

	void start() {
		hipEventRecord(start_event, 0);
	}

	float count_time() {
		float elapsed_time;
		hipEventRecord(end_event, 0);
		hipEventSynchronize(end_event);
		hipEventElapsedTime(&elapsed_time, start_event, end_event);
		return elapsed_time;
	}
};

double* fill_matrix_rnd(size_t matrix_size) {

	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<double> distrib(VALUES_MIN, VALUES_MAX);
	size_t element_count = matrix_size * matrix_size;
	double* result = new double[element_count];
	for (size_t i = 0; i < element_count; ++i) {
		result[i] = distrib(gen);
	}
	return result;
}

__global__ void mul_on_gpu_shared_kernel(double* a, double* b, double* result, size_t matrix_size) {
	size_t tx = threadIdx.x;
	size_t ty = threadIdx.y;

	size_t i = blockDim.y * blockIdx.y + ty;
	size_t j = blockDim.x * blockIdx.x + tx;

	size_t aj;
	size_t bi;
	double sum = 0.0;
	
	for (size_t ind = 0; ind * CUDA_BLOCK_SIZE < matrix_size; ind++) {
		aj = tx + CUDA_BLOCK_SIZE * ind;
		bi = ty + CUDA_BLOCK_SIZE * ind;

		__shared__ double as[CUDA_BLOCK_SIZE][CUDA_BLOCK_SIZE];
		__shared__ double bs[CUDA_BLOCK_SIZE][CUDA_BLOCK_SIZE];

		as[ty][tx] = 0;
		bs[ty][tx] = 0;
		if (i < matrix_size && aj < matrix_size)
		{
			as[ty][tx] = a[i * matrix_size + aj];
		}
		if (j < matrix_size && bi < matrix_size)
		{
			bs[ty][tx] = b[bi * matrix_size + j];
		}

		__syncthreads();
		for (size_t k = 0; k < CUDA_BLOCK_SIZE; k++)
			sum += as[ty][k] * bs[k][tx];
		__syncthreads();
	}

	if (i < matrix_size && j < matrix_size)
	{
		result[i * matrix_size + j] = sum;
	}	
}

__global__ void mul_on_gpu_kernel(double* a, double* b, double* result, size_t matrix_size) {
	size_t i = blockDim.y * blockIdx.y + threadIdx.y;
	size_t j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= matrix_size || j >= matrix_size)
		return;

	size_t ind = i * matrix_size + j;
	result[ind] = 0;

	for (size_t k = 0; k < matrix_size; ++k) {
		result[ind] += a[i * matrix_size + k] * b[k * matrix_size + j];
	}
}


double process_on_gpu(double* matrix_A, double* matrix_B, double* result, size_t matrix_size, MultType mult_type) {
	MyCudaTimer timer;

	double* gpu_mem_A;
	double* gpu_mem_B;
	double* gpu_mem_res;
	size_t bytes_count = matrix_size * matrix_size * sizeof(double);
	hipMalloc((void**)&gpu_mem_A, bytes_count);
	hipMalloc((void**)&gpu_mem_B, bytes_count);
	hipMalloc((void**)&gpu_mem_res, bytes_count);

	hipMemcpy(gpu_mem_A, matrix_A, bytes_count, hipMemcpyHostToDevice);
	hipMemcpy(gpu_mem_B, matrix_B, bytes_count, hipMemcpyHostToDevice);

	dim3 cuda_threads(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);
	dim3 cuda_blocks((matrix_size + cuda_threads.x - 1) / cuda_threads.x, (matrix_size + cuda_threads.y - 1) / cuda_threads.y);

	timer.start();
	switch (mult_type)
	{
	case MultType::GPU:
		mul_on_gpu_kernel <<< cuda_blocks, cuda_threads >>> (gpu_mem_A, gpu_mem_B, gpu_mem_res, matrix_size);
		break;
	case MultType::GPU_SHARED:
		mul_on_gpu_shared_kernel <<< cuda_blocks, cuda_threads >>> (gpu_mem_A, gpu_mem_B, gpu_mem_res, matrix_size);
		break;
	default:
		return -1;
	}
	

	float elapsed_time = timer.count_time();

	hipMemcpy(result, gpu_mem_res, bytes_count, hipMemcpyDeviceToHost);

	hipFree(gpu_mem_A);
	hipFree(gpu_mem_B);
	hipFree(gpu_mem_res);

	return elapsed_time / 1000.0f;
}

double get_max_diff(double* matrix_A, double* matrix_B, size_t matrix_size) {
	size_t element_count = matrix_size * matrix_size;
	double result = 0.0;
	for (size_t i = 0; i < element_count; ++i) {

		result = std::max(result, std::fabs(matrix_A[i] - matrix_B[i]));
	}

	return result;
}


int main(int argc, char* argv[]) {

	for (size_t matrix_size : MATRIX_SIZES_TO_TEST) {
		double* matrix_A = fill_matrix_rnd(matrix_size);
		double* matrix_B = fill_matrix_rnd(matrix_size);
		double* res_on_gpu = new double[matrix_size * matrix_size];
		double* res_on_gpu_shared = new double[matrix_size * matrix_size];

		float time_on_gpu = process_on_gpu(matrix_A, matrix_B, res_on_gpu, matrix_size, MultType::GPU);
		float time_on_gpu_shared = process_on_gpu(matrix_A, matrix_B, res_on_gpu_shared, matrix_size, MultType::GPU_SHARED);
		double max_diff = get_max_diff(res_on_gpu, res_on_gpu_shared, matrix_size);

		std::cout << "-------------------------------" << std::endl;
		std::cout << "matrix_size: \t" << matrix_size << std::endl;
		std::cout << "time on GPU: \t" << time_on_gpu << std::endl;
		std::cout << "time on GPU_shared: \t" << time_on_gpu_shared << std::endl;
		std::cout << "max diff: \t" << max_diff << std::endl;
		std::cout << std::endl;

		delete[] matrix_A;
		delete[] matrix_B;
		delete[] res_on_gpu;
		delete[] res_on_gpu_shared;
	}
	return 0;
}